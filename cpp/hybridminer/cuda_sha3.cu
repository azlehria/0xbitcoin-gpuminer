#include "hip/hip_runtime.h"
// default magic numbers
#define INTENSITY 23
#define CUDA_DEVICE 0
// default magic numbers

#include <stdio.h>
#include <stdint.h>
#include <inttypes.h>
#include <time.h>
#include <sys/timeb.h>

#if defined(_MSC_VER)
#  include <process.h>
#else
#  include <sys/types.h>
#  include <unistd.h>
#endif

#include "cudasolver.h"

/*
Author: Mikers
date march 4, 2018 for 0xbitcoin dev

based off of https://github.com/Dunhili/SHA3-gpu-brute-force-cracker/blob/master/sha3.cu

 * Author: Brian Bowden
 * Date: 5/12/14
 *
 * This is the parallel version of SHA-3.
 */

#ifdef __INTELLISENSE__
 /* reduce vstudio warnings (__byteperm, blockIdx...) */
#include <hip/device_functions.h>
#include <>
#define __launch_bounds__(max_tpb, min_blocks)
#endif

#define TPB52 1024
#define TPB50 384
#define NPT 2
#define NBN 2

int32_t intensity;
int32_t cuda_device;
int32_t clock_speed;
int32_t compute_version;
int32_t h_done[1] = { 0 };
struct timeb start, end;

uint64_t cnt;
uint64_t printable_hashrate_cnt;
uint64_t print_counter;

bool gpu_initialized;
bool new_input;

uint8_t solution[32] = { 0 };
uint8_t* h_message;
uint8_t init_message[84];

int32_t* d_done;
uint64_t* d_solution;

uint8_t* d_challenge;
uint8_t* d_hash_prefix;
__constant__ uint64_t d_mid[25];

#define ROTL64(x, y) (((x) << (y)) | ((x) >> (64 - (y))))

__device__ __constant__ const uint64_t RC[24] = {
  0x0000000000000001, 0x0000000000008082, 0x800000000000808a,
  0x8000000080008000, 0x000000000000808b, 0x0000000080000001,
  0x8000000080008081, 0x8000000000008009, 0x000000000000008a,
  0x0000000000000088, 0x0000000080008009, 0x000000008000000a,
  0x000000008000808b, 0x800000000000008b, 0x8000000000008089,
  0x8000000000008003, 0x8000000000008002, 0x8000000000000080,
  0x000000000000800a, 0x800000008000000a, 0x8000000080008081,
  0x8000000000008080, 0x0000000080000001, 0x8000000080008008
};

__device__ __forceinline__
uint64_t bswap_64( uint64_t x )
{
	return ((uint64_t)(__byte_perm((uint32_t) x, 0, 0x0123)) << 32)
		   + __byte_perm((uint32_t)(x >> 32), 0, 0x0123);
}

__device__ __forceinline__
uint64_t xor5( uint64_t a, uint64_t b, uint64_t c, uint64_t d, uint64_t e )
{
  uint64_t output;
  asm( "xor.b64 %0, %1, %2;" : "=l"(output) : "l"(d) ,"l"(e) );
  asm( "xor.b64 %0, %0, %1;" : "+l"(output) : "l"(c) );
  asm( "xor.b64 %0, %0, %1;" : "+l"(output) : "l"(b) );
  asm( "xor.b64 %0, %0, %1;" : "+l"(output) : "l"(a) );
  return output;
}

__device__
bool keccak( uint64_t nounce, uint64_t target )
{
  uint64_t state[25], C[5], D[5];

  int32_t x;

  state[ 2] = d_mid[ 2] ^ ROTL64(nounce, 44);
  state[ 4] = d_mid[ 4] ^ ROTL64(nounce, 14);

  state[ 6] = d_mid[ 6] ^ ROTL64(nounce, 20);
  state[ 9] = d_mid[ 9] ^ ROTL64(nounce, 62);

  state[11] = d_mid[11] ^ ROTL64(nounce, 7);
  state[13] = d_mid[13] ^ ROTL64(nounce, 8);

  state[15] = d_mid[15] ^ ROTL64(nounce, 27);
  state[18] = d_mid[18] ^ ROTL64(nounce, 16);

  state[20] = d_mid[20] ^ ROTL64(nounce, 63);
  state[21] = d_mid[21] ^ ROTL64(nounce, 55);
  state[22] = d_mid[22] ^ ROTL64(nounce, 39);

  //  Chi
  // for j = 0 to 25, j += 5
  //     for i = 0 to 5
  //         C[i] = state[j + i];
  //     for i = 0 to 5
  //         state[j + 1] ^= (~C[(i + 1) % 5]) & C[(i + 2) % 5];
  state[ 0] = d_mid[ 0] ^ ( ~d_mid[ 1] ) & state[ 2];
  state[ 1] = d_mid[ 1] ^ ( ~state[ 2] ) & d_mid[ 3];
  state[ 2] = state[ 2] ^ ( ~d_mid[ 3] ) & state[ 4];
  state[ 3] = d_mid[ 3] ^ ( ~state[ 4] ) & d_mid[ 0];
  state[ 4] = state[ 4] ^ ( ~d_mid[ 0] ) & d_mid[ 1];

  C[0] = state[ 6];
  state[ 5] = d_mid[ 5] ^ ( ~state[ 6] ) & d_mid[7];
  state[ 6] = state[ 6] ^ ( ~d_mid[ 7] ) & d_mid[8];
  state[ 7] = d_mid[ 7] ^ ( ~d_mid[ 8] ) & state[9];
  state[ 8] = d_mid[ 8] ^ ( ~state[ 9] ) & d_mid[5];
  state[ 9] = state[ 9] ^ ( ~d_mid[ 5] ) & C[0];

  C[0] = state[11];
  state[10] = d_mid[10] ^ ( ~state[11] ) & d_mid[12];
  state[11] = state[11] ^ ( ~d_mid[12] ) & state[13];
  state[12] = d_mid[12] ^ ( ~state[13] ) & d_mid[14];
  state[13] = state[13] ^ ( ~d_mid[14] ) & d_mid[10];
  state[14] = d_mid[14] ^ ( ~d_mid[10] ) & C[0];

  C[0] = state[15];
  state[15] = state[15] ^ ( ~d_mid[16] ) & d_mid[17];
  state[16] = d_mid[16] ^ ( ~d_mid[17] ) & state[18];
  state[17] = d_mid[17] ^ ( ~state[18] ) & d_mid[19];
  state[18] = state[18] ^ ( ~d_mid[19] ) & C[0];
  state[19] = d_mid[19] ^ ( ~C[0] ) & d_mid[16];

  C[0] = state[20];
  C[1] = state[21];
  state[20] = state[20] ^ ( ~state[21] ) & state[22];
  state[21] = state[21] ^ ( ~state[22] ) & d_mid[23];
  state[22] = state[22] ^ ( ~d_mid[23] ) & d_mid[24];
  state[23] = d_mid[23] ^ ( ~d_mid[24] ) & C[0];
  state[24] = d_mid[24] ^ ( ~C[0] ) & C[1];

  //  Iota
  state[0] ^= RC[0];

#if __CUDA_ARCH__ >= 600
#pragma unroll 22
#endif
  for( int32_t i = 1; i < 23; i++ )
  {
    // Theta
    // for i = 0 to 5
    //    C[i] = state[i] ^ state[i + 5] ^ state[i + 10] ^ state[i + 15] ^ state[i + 20];
    for (x = 0; x < 5; x++) {
      C[x] = xor5( state[x], state[x + 5], state[x + 10], state[x + 15], state[x + 20] );
    }

    // for i = 0 to 5
    //     temp = C[(i + 4) % 5] ^ ROTL64(C[(i + 1) % 5], 1);
    //     for j = 0 to 25, j += 5
    //          state[j + i] ^= temp;
#if __CUDA_ARCH__ >= 600
    D[0] = ROTL64(C[1], 1) ^ C[4];
    D[1] = ROTL64(C[2], 1) ^ C[0];
    D[2] = ROTL64(C[3], 1) ^ C[1];
    D[3] = ROTL64(C[4], 1) ^ C[2];
    D[4] = ROTL64(C[0], 1) ^ C[3];

    for (x = 0; x < 5; x++) {
      state[x]      ^= D[x];
      state[x + 5]  ^= D[x];
      state[x + 10] ^= D[x];
      state[x + 15] ^= D[x];
      state[x + 20] ^= D[x];
    }
#else
    D[0] = ROTL64(C[1], 1) ^ C[4];
    state[ 0] ^= D[0];
    state[ 5] ^= D[0];
    state[10] ^= D[0];
    state[15] ^= D[0];
    state[20] ^= D[0];

    D[0] = ROTL64(C[2], 1) ^ C[0];
    state[ 1] ^= D[0];
    state[ 6] ^= D[0];
    state[11] ^= D[0];
    state[16] ^= D[0];
    state[21] ^= D[0];

    D[0] = ROTL64(C[3], 1) ^ C[1];
    state[ 2] ^= D[0];
    state[ 7] ^= D[0];
    state[12] ^= D[0];
    state[17] ^= D[0];
    state[22] ^= D[0];

    D[0] = ROTL64(C[4], 1) ^ C[2];
    state[ 3] ^= D[0];
    state[ 8] ^= D[0];
    state[13] ^= D[0];
    state[18] ^= D[0];
    state[23] ^= D[0];

    D[0] = ROTL64(C[0], 1) ^ C[3];
    state[ 4] ^= D[0];
    state[ 9] ^= D[0];
    state[14] ^= D[0];
    state[19] ^= D[0];
    state[24] ^= D[0];
#endif

    // Rho Pi
    // for i = 0 to 24
    //     j = piln[i];
    //     C[0] = state[j];
    //     state[j] = ROTL64(temp, r[i]);
    //     temp = C[0];
    C[0] = state[1];
    state[ 1] = ROTL64( state[ 6], 44 );
    state[ 6] = ROTL64( state[ 9], 20 );
    state[ 9] = ROTL64( state[22], 61 );
    state[22] = ROTL64( state[14], 39 );
    state[14] = ROTL64( state[20], 18 );
    state[20] = ROTL64( state[ 2], 62 );
    state[ 2] = ROTL64( state[12], 43 );
    state[12] = ROTL64( state[13], 25 );
    state[13] = ROTL64( state[19],  8 );
    state[19] = ROTL64( state[23], 56 );
    state[23] = ROTL64( state[15], 41 );
    state[15] = ROTL64( state[ 4], 27 );
    state[ 4] = ROTL64( state[24], 14 );
    state[24] = ROTL64( state[21],  2 );
    state[21] = ROTL64( state[ 8], 55 );
    state[ 8] = ROTL64( state[16], 45 );
    state[16] = ROTL64( state[ 5], 36 );
    state[ 5] = ROTL64( state[ 3], 28 );
    state[ 3] = ROTL64( state[18], 21 );
    state[18] = ROTL64( state[17], 15 );
    state[17] = ROTL64( state[11], 10 );
    state[11] = ROTL64( state[ 7],  6 );
    state[ 7] = ROTL64( state[10],  3 );
    state[10] = ROTL64( C[0], 1 );

    //  Chi
    // for j = 0 to 25, j += 5
    //     for i = 0 to 5
    //         C[i] = state[j + i];
    //     for i = 0 to 5
    //         state[j + 1] ^= (~C[(i + 1) % 5]) & C[(i + 2) % 5];
    C[0] = state[ 0];
    C[1] = state[ 1];
    state[ 0] ^= ( ~state[1] ) & state[2];
    state[ 1] ^= ( ~state[2] ) & state[3];
    state[ 2] ^= ( ~state[3] ) & state[4];
    state[ 3] ^= ( ~state[4] ) & C[0];
    state[ 4] ^= ( ~C[0] ) & C[1];

    C[0] = state[ 5];
    C[1] = state[ 6];
    state[ 5] ^= ( ~state[6] ) & state[7];
    state[ 6] ^= ( ~state[7] ) & state[8];
    state[ 7] ^= ( ~state[8] ) & state[9];
    state[ 8] ^= ( ~state[9] ) & C[0];
    state[ 9] ^= ( ~C[0] ) & C[1];

    C[0] = state[10];
    C[1] = state[11];
    state[10] ^= ( ~state[11] ) & state[12];
    state[11] ^= ( ~state[12] ) & state[13];
    state[12] ^= ( ~state[13] ) & state[14];
    state[13] ^= ( ~state[14] ) & C[0];
    state[14] ^= ( ~C[0] ) & C[1];

    C[0] = state[15];
    C[1] = state[16];
    state[15] ^= ( ~state[16] ) & state[17];
    state[16] ^= ( ~state[17] ) & state[18];
    state[17] ^= ( ~state[18] ) & state[19];
    state[18] ^= ( ~state[19] ) & C[0];
    state[19] ^= ( ~C[0] ) & C[1];

    C[0] = state[20];
    C[1] = state[21];
    state[20] ^= ( ~state[21] ) & state[22];
    state[21] ^= ( ~state[22] ) & state[23];
    state[22] ^= ( ~state[23] ) & state[24];
    state[23] ^= ( ~state[24] ) & C[0];
    state[24] ^= ( ~C[0] ) & C[1];

    //  Iota
    state[0] ^= RC[i];
  }
  for (x = 0; x < 5; x++) {
    C[x] = xor5( state[x], state[x + 5], state[x + 10], state[x + 15], state[x + 20] );
  }

  state[ 0] ^= ROTL64(C[1], 1) ^ C[4];
  state[ 6] ^= ROTL64(C[2], 1) ^ C[0];
  state[12] ^= ROTL64(C[3], 1) ^ C[1];

  state[ 1] = ROTL64( state[ 6], 44 );
  state[ 2] = ROTL64( state[12], 43 );

  state[ 0] ^= ( ~state[1] ) & state[2];

  state[0] ^= RC[23];

  return bswap_64( state[0] ) <= target;
  // memcpy( output, state, 32 );
}

// hash length is 256 bits
#if __CUDA_ARCH__ > 500
__global__ __launch_bounds__( TPB52, 1 )
#else
__global__ __launch_bounds__( TPB50, 2 )
#endif
void gpu_mine( uint64_t* solution, int32_t* done, uint64_t cnt, uint32_t threads, uint64_t target )
{
  uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

#if __CUDA_ARCH__ > 500
  uint64_t step = gridDim.x * blockDim.x;
  uint64_t maxNonce = cnt + threads;
  for( uint64_t nounce = cnt + thread; nounce < maxNonce; nounce += step )
  {
#else
  uint32_t nounce = cnt + thread;
  if( thread < threads )
  {
#endif

    if( keccak( nounce, target ) )
    {
	  const uint32_t temp = atomicExch( &done[0], thread );
      if( done[0] == thread )
      {
        *solution = nounce;
      }
      return;
    }
  }
}

__host__
void stop_solving()
{
  h_done[0] = 0xff;
}

__host__
int32_t gcd( int32_t a, int32_t b )
{
  return ( a == 0 ) ? b : gcd( b % a, a );
}

__host__
uint64_t getHashCount()
{
  return cnt;
}
__host__
void resetHashCount()
{
  cnt = 0;
  //printable_hashrate_cnt = 0;
}

__host__
void send_to_device( uint64_t* message )
{
	uint64_t C[4], D[5], mid[25];
	C[0] = message[0] ^ message[5] ^ message[10] ^ 0x100000000ull;
	C[1] = message[1] ^ message[6] ^ 0x8000000000000000ull;
	C[2] = message[2] ^ message[7];
	C[3] = message[4] ^ message[9];

	D[0] = ROTL64(C[1], 1) ^ C[3];
	D[1] = ROTL64(C[2], 1) ^ C[0];
	D[2] = ROTL64(message[3], 1) ^ C[1];
	D[3] = ROTL64(C[3], 1) ^ C[2];
	D[4] = ROTL64(C[0], 1) ^ message[3];

	mid[ 0] = message[ 0] ^ D[0];//S[0]
	mid[ 1] = ROTL64( message[6] ^ D[1], 44 );//S[1]
	mid[ 2] = ROTL64(D[2], 43);//D[2]
	mid[ 3] = ROTL64(D[3], 21);//D[0]
	mid[ 4] = ROTL64(D[4], 14);
	mid[ 5] = ROTL64( message[3] ^ D[3], 28 );//S[5]
	mid[ 6] = ROTL64( message[9] ^ D[4], 20 );//S[6]
	mid[ 7] = ROTL64( message[10] ^ D[0] ^ 0x100000000ull, 3 );//S[7]
	mid[ 8] = ROTL64( 0x8000000000000000ull ^ D[1], 45 );//S[8]
	mid[ 9] = ROTL64(D[2], 61);
	mid[10] = ROTL64( message[1] ^ D[1],  1 );//S[10]
	mid[11] = ROTL64( message[7] ^ D[2],  6 );//S[11]
	mid[12] = ROTL64(D[3], 25);
	mid[13] = ROTL64(D[4],  8);
	mid[14] = ROTL64(D[0], 18);
	mid[15] = ROTL64( message[4] ^ D[4], 27 );//S[15]
	mid[16] = ROTL64( message[5] ^ D[0], 36 );//S[16]
	mid[17] = ROTL64(D[1], 10);
	mid[18] = ROTL64(D[2], 15);
	mid[19] = ROTL64(D[3], 56);
	mid[20] = ROTL64( message[2] ^ D[2], 62 );//S[20]
	mid[21] = ROTL64(D[3], 55);
	mid[22] = ROTL64(D[4], 39);
	mid[23] = ROTL64(D[0], 41);
	mid[24] = ROTL64(D[1],  2);

	hipMemcpyToSymbol(HIP_SYMBOL( d_mid), mid, sizeof( mid ), cuda_device, hipMemcpyHostToDevice);
}

/**
 * Initializes the global variables by calling the hipGetDeviceProperties().
 */
__host__
void gpu_init()
{
  hipDeviceProp_t device_prop;
  int32_t device_count;
  ftime( &start );
  
  srand((time(NULL) & 0xFFFF) | (getpid() << 16));

  char config[10];
  FILE * inf;
  inf = fopen( "0xbtc.conf", "r" );
  if( inf )
  {
    fgets( config, 10, inf );
    fclose( inf );
    intensity = atol( strtok( config, " " ) );
    cuda_device = atol( strtok( NULL, " " ) );
  }
  else
  {
    intensity = INTENSITY;
    cuda_device = CUDA_DEVICE;
  }

  hipGetDeviceCount( &device_count );

  if( hipGetDeviceProperties( &device_prop, cuda_device ) != hipSuccess )
  {
    printf( "Problem getting properties for device, exiting...\n" );
    exit( EXIT_FAILURE );
  }

  hipSetDevice( cuda_device );

  if( !gpu_initialized )
  {
    hipDeviceReset();
    hipSetDeviceFlags( hipDeviceScheduleBlockingSync );

    hipMalloc( (void**)&d_done, sizeof( int32_t ) );
    hipMalloc( (void**)&d_solution, 32 ); // solution
    hipHostMalloc( (void**)&h_message, 32 );

    (uint32_t&)(init_message[52]) = 014533075101u;
    (uint32_t&)(init_message[56]) = 014132271150u;
    for(int8_t i_rand = 60; i_rand < 84; i_rand++){
      init_message[i_rand] = (uint8_t)rand() % 256;
    }
	memcpy( solution, &init_message[52], 32 );

    gpu_initialized = true;
  }

  compute_version = device_prop.major * 100 + device_prop.minor * 10;

  // convert from GHz to hertz
  clock_speed = (int32_t)( device_prop.memoryClockRate * 1000 * 1000 );

  //cnt = 0;
  printable_hashrate_cnt = 0;
  print_counter = 0;

  if( new_input ) new_input = false;
}

__host__
void update_mining_inputs()
{
  new_input = true;
}

__host__
bool find_message( uint64_t target, uint8_t * hash_prefix )
{
  h_done[0] = 0;
  if( !gpu_initialized )
  {
    gpu_init();
  }
  new_input = false;

  for(int8_t i = 0; i < 52; i++){
    init_message[i] = hash_prefix[i];
  }
  send_to_device( (uint64_t*)init_message );

  hipMemcpy( d_done, h_done, sizeof( int32_t ), hipMemcpyHostToDevice );
  hipMemset( d_solution, 0xff, 32 );

  uint32_t threads = 1UL << intensity;

  uint32_t tpb;
  dim3 grid;
  if( compute_version > 500 )
  {
    tpb = TPB52;
    grid.x = ( threads + ( NPT*tpb ) - 1 ) / ( NPT*tpb );
  }
  else
  {
    tpb = TPB50;
    grid.x = ( threads + tpb - 1 ) / tpb;
  }
  const dim3 block( tpb );

  gpu_mine <<< grid, block >>> ( d_solution, d_done, cnt, threads, target );
  // hipError_t cudaerr = hipDeviceSynchronize();
  // if( cudaerr != hipSuccess )
  // {
  //  printf( "kernel launch failed with error %d: \x1b[38;5;196m%s.\x1b[0m\n", cudaerr, hipGetErrorString( cudaerr ) );
  //  exit( EXIT_FAILURE );
  // }

  if( h_done[0] < 0 ) return false;

  cnt += threads;
  printable_hashrate_cnt += threads;

  hipMemcpy( h_done, d_done, sizeof( int32_t ), hipMemcpyDeviceToHost );
  hipMemcpy( h_message, d_solution, 8, hipMemcpyDeviceToHost );
  memcpy( &solution[12], h_message, 8 );

  ftime( &end );
  double t = (double)((end.time * 1000 + end.millitm) - (start.time * 1000 + start.millitm)) / 1000;

  if( t*10 > print_counter )
  {
    print_counter++;

	// maybe breaking the control codes into macros is a good idea . . .
	printf( "\x1b[s\x1b[3;67f\x1b[38;5;221m%*.2f\x1b[0m\x1b[u"
			"\x1b[s\x1b[3;29f\x1b[38;5;208m%*" PRIu64 "\x1b[0m\x1b[u",
			8, ( (double)printable_hashrate_cnt / t / 1000000 ),
		  26, printable_hashrate_cnt );
  }

  return ( h_done[0] > 0 );
}

__host__
void gpu_cleanup()
{
  if( !gpu_initialized ) return;

  hipDeviceSynchronize();

  hipFree( d_done );
  hipFree( d_solution );
  hipHostFree( h_message );
}
