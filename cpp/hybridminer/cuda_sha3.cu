#include "hip/hip_runtime.h"
// default magic numbers
#define INTENSITY 23
#define CUDA_DEVICE 0
// default magic numbers

#include <stdio.h>
#include <stdint.h>
#include <inttypes.h>
#include <time.h>
#include <sys/timeb.h>

#if defined(_MSC_VER)
#  include <process.h>
#else
#  include <sys/types.h>
#  include <unistd.h>
#endif

#include "cudasolver.h"

/*
Author: Mikers
date march 4, 2018 for 0xbitcoin dev

based off of https://github.com/Dunhili/SHA3-gpu-brute-force-cracker/blob/master/sha3.cu

 * Author: Brian Bowden
 * Date: 5/12/14
 *
 * This is the parallel version of SHA-3.
 */

#ifdef __INTELLISENSE__
 /* reduce vstudio warnings (__byteperm, blockIdx...) */
#include <hip/device_functions.h>
#include <>
#define __launch_bounds__(max_tpb, min_blocks)
#endif

#define TPB52 1024
#define TPB50 384
#define NPT 2
#define NBN 2

int32_t intensity;
int32_t cuda_device;
int32_t clock_speed;
int32_t compute_version;
int32_t h_done[1] = { 0 };
struct timeb start, end;

uint64_t cnt;
uint64_t printable_hashrate_cnt;
uint64_t print_counter;

bool gpu_initialized;
bool new_input;

uint8_t solution[32] = { 0 };
uint8_t* h_message;
uint8_t init_message[84];

int32_t* d_done;
uint64_t* d_solution;

uint8_t* d_challenge;
uint8_t* d_hash_prefix;
__constant__ uint64_t d_mid[25];
__constant__ uint64_t d_target;

#define ROTL64(x, y) (((x) << (y)) ^ ((x) >> (64 - (y))))

__device__ __constant__ const uint64_t RC[24] = {
  0x0000000000000001, 0x0000000000008082, 0x800000000000808a,
  0x8000000080008000, 0x000000000000808b, 0x0000000080000001,
  0x8000000080008081, 0x8000000000008009, 0x000000000000008a,
  0x0000000000000088, 0x0000000080008009, 0x000000008000000a,
  0x000000008000808b, 0x800000000000008b, 0x8000000000008089,
  0x8000000000008003, 0x8000000000008002, 0x8000000000000080,
  0x000000000000800a, 0x800000008000000a, 0x8000000080008081,
  0x8000000000008080, 0x0000000080000001, 0x8000000080008008
};

__device__ __forceinline__
uint64_t bswap_64( uint64_t x )
{
  return ((uint64_t)(__byte_perm((uint32_t) x, 0, 0x0123)) << 32)
    ^ __byte_perm((uint32_t)(x >> 32), 0, 0x0123);
}

__device__ __forceinline__
uint64_t xor5( uint64_t a, uint64_t b, uint64_t c, uint64_t d, uint64_t e )
{
  uint64_t output = 0;
  asm( "{"
       "  xor.b64 %0, %1, %2;"
       "  xor.b64 %0, %0, %3;"
       "  xor.b64 %0, %0, %4;"
       "  xor.b64 %0, %0, %5;"
       "}" : "+l"(output) : "l"(a), "l"(b), "l"(c), "l"(d), "l"(e) );
  return output;
}

__device__ __forceinline__
uint64_t chi( uint64_t a, uint64_t b, uint64_t c )
{
#if __CUDA_ARCH__ >= 500 && CUDA_VERSION >= 7050
  uint64_t output = 0;
  asm( "{"
       "  lop3.b32 [&output], [&a], [&b], [&c], 0xD2;"
       "  lop3.b32 [&output+4], [&a+4], [&b+4], [&c+4], 0xD2;"
       "}" );
  return output;
#else
  return a ^ ((~b) & c);
#endif
}

__device__
bool keccak( uint64_t const nounce )
{
  uint64_t state[25], C[5], D[5];

  int32_t x;

  state[ 2] = d_mid[ 2] ^ ROTL64(nounce, 44);
  state[ 4] = d_mid[ 4] ^ ROTL64(nounce, 14);
  state[ 0] = chi( d_mid[ 0], d_mid[ 1], state[ 2] );
  state[ 0] = state[0] ^ RC[0];
  state[ 1] = chi( d_mid[ 1], state[ 2], d_mid[ 3] );
  state[ 2] = chi( state[ 2], d_mid[ 3], state[ 4] );
  state[ 3] = chi( d_mid[ 3], state[ 4], d_mid[ 0] );
  state[ 4] = chi( state[ 4], d_mid[ 0], d_mid[ 1] );

  C[0] = d_mid[ 6] ^ ROTL64(nounce, 20);
  state[ 9] = d_mid[ 9] ^ ROTL64(nounce, 62);
  state[ 5] = chi( d_mid[ 5], C[ 0], d_mid[7] );
  state[ 6] = chi( C[ 0], d_mid[ 7], d_mid[8] );
  state[ 7] = chi( d_mid[ 7], d_mid[ 8], state[9] );
  state[ 8] = chi( d_mid[ 8], state[ 9], d_mid[5] );
  state[ 9] = chi( state[ 9], d_mid[ 5], C[0] );

  C[0] = d_mid[11] ^ ROTL64(nounce, 7);
  state[13] = d_mid[13] ^ ROTL64(nounce, 8);
  state[10] = chi( d_mid[10], C[0], d_mid[12] );
  state[11] = chi( C[0], d_mid[12], state[13] );
  state[12] = chi( d_mid[12], state[13], d_mid[14] );
  state[13] = chi( state[13], d_mid[14], d_mid[10] );
  state[14] = chi( d_mid[14], d_mid[10], C[0] );

  C[0] = d_mid[15] ^ ROTL64(nounce, 27);
  state[18] = d_mid[18] ^ ROTL64(nounce, 16);
  state[15] = chi( C[0], d_mid[16], d_mid[17] );
  state[16] = chi( d_mid[16], d_mid[17], state[18] );
  state[17] = chi( d_mid[17], state[18], d_mid[19] );
  state[18] = chi( state[18], d_mid[19], C[0] );
  state[19] = chi( d_mid[19], C[0], d_mid[16] );

  C[0] = d_mid[20] ^ ROTL64(nounce, 63);
  C[1] = d_mid[21] ^ ROTL64(nounce, 55);
  state[22] = d_mid[22] ^ ROTL64(nounce, 39);
  state[20] = chi( C[0], C[1], state[22] );
  state[21] = chi( C[1], state[22], d_mid[23] );
  state[22] = chi( state[22], d_mid[23], d_mid[24] );
  state[23] = chi( d_mid[23], d_mid[24], C[0] );
  state[24] = chi( d_mid[24], C[0], C[1] );

#if __CUDA_ARCH__ >= 600
#pragma unroll 22
#endif
  for( int32_t i = 1; i < 23; ++i )
  {
    // Theta
    // for i = 0 to 5
    //    C[i] = state[i] ^ state[i + 5] ^ state[i + 10] ^ state[i + 15] ^ state[i + 20];
    for( x = 0; x < 5; ++x )
    {
      C[x] = xor5( state[x], state[x + 5], state[x + 10], state[x + 15], state[x + 20] );
    }

    // for i = 0 to 5
    //     temp = C[(i + 4) % 5] ^ ROTL64(C[(i + 1) % 5], 1);
    //     for j = 0 to 25, j += 5
    //          state[j + i] ^= temp;
#if __CUDA_ARCH__ >= 600
    D[0] = ROTL64(C[1], 1) ^ C[4];
    D[1] = ROTL64(C[2], 1) ^ C[0];
    D[2] = ROTL64(C[3], 1) ^ C[1];
    D[3] = ROTL64(C[4], 1) ^ C[2];
    D[4] = ROTL64(C[0], 1) ^ C[3];

    for (x = 0; x < 5; ++x)
    {
      // As of 9.1, compiler still isn't smart enough to realize `a ^= b` == `a = a ^ b`
      // The latter form is fractionally faster
      state[x]      = state[x]      ^ D[x];
      state[x +  5] = state[x +  5] ^ D[x];
      state[x + 10] = state[x + 10] ^ D[x];
      state[x + 15] = state[x + 15] ^ D[x];
      state[x + 20] = state[x + 20] ^ D[x];
    }
#else
    D[0] = ROTL64(C[1], 1) ^ C[4];
    state[ 0] = state[ 0] ^ D[0];
    state[ 5] = state[ 5] ^ D[0];
    state[10] = state[10] ^ D[0];
    state[15] = state[15] ^ D[0];
    state[20] = state[20] ^ D[0];

    D[0] = ROTL64(C[2], 1) ^ C[0];
    state[ 1] = state[ 1] ^ D[0];
    state[ 6] = state[ 6] ^ D[0];
    state[11] = state[11] ^ D[0];
    state[16] = state[16] ^ D[0];
    state[21] = state[21] ^ D[0];

    D[0] = ROTL64(C[3], 1) ^ C[1];
    state[ 2] = state[ 2] ^ D[0];
    state[ 7] = state[ 7] ^ D[0];
    state[12] = state[12] ^ D[0];
    state[17] = state[17] ^ D[0];
    state[22] = state[22] ^ D[0];

    D[0] = ROTL64(C[4], 1) ^ C[2];
    state[ 3] = state[ 3] ^ D[0];
    state[ 8] = state[ 8] ^ D[0];
    state[13] = state[13] ^ D[0];
    state[18] = state[18] ^ D[0];
    state[23] = state[23] ^ D[0];

    D[0] = ROTL64(C[0], 1) ^ C[3];
    state[ 4] = state[ 4] ^ D[0];
    state[ 9] = state[ 9] ^ D[0];
    state[14] = state[14] ^ D[0];
    state[19] = state[19] ^ D[0];
    state[24] = state[24] ^ D[0];
#endif

    // Rho Pi
    // for i = 0 to 24
    //     j = piln[i];
    //     C[0] = state[j];
    //     state[j] = ROTL64(temp, r[i]);
    //     temp = C[0];
    C[0] = state[1];
    state[ 1] = ROTL64( state[ 6], 44 );
    state[ 6] = ROTL64( state[ 9], 20 );
    state[ 9] = ROTL64( state[22], 61 );
    state[22] = ROTL64( state[14], 39 );
    state[14] = ROTL64( state[20], 18 );
    state[20] = ROTL64( state[ 2], 62 );
    state[ 2] = ROTL64( state[12], 43 );
    state[12] = ROTL64( state[13], 25 );
    state[13] = ROTL64( state[19],  8 );
    state[19] = ROTL64( state[23], 56 );
    state[23] = ROTL64( state[15], 41 );
    state[15] = ROTL64( state[ 4], 27 );
    state[ 4] = ROTL64( state[24], 14 );
    state[24] = ROTL64( state[21],  2 );
    state[21] = ROTL64( state[ 8], 55 );
    state[ 8] = ROTL64( state[16], 45 );
    state[16] = ROTL64( state[ 5], 36 );
    state[ 5] = ROTL64( state[ 3], 28 );
    state[ 3] = ROTL64( state[18], 21 );
    state[18] = ROTL64( state[17], 15 );
    state[17] = ROTL64( state[11], 10 );
    state[11] = ROTL64( state[ 7],  6 );
    state[ 7] = ROTL64( state[10],  3 );
    state[10] = ROTL64( C[0], 1 );

    //  Chi
    // for j = 0 to 25, j += 5
    //     for i = 0 to 5
    //         C[i] = state[j + i];
    //     for i = 0 to 5
    //         state[j + 1] ^= (~C[(i + 1) % 5]) & C[(i + 2) % 5];
    C[0] = state[ 0];
    C[1] = state[ 1];
    state[ 0] = chi( state[ 0], state[1], state[2] ) ^ RC[i];
    state[ 1] = chi( state[ 1], state[2], state[3] );
    state[ 2] = chi( state[ 2], state[3], state[4] );
    state[ 3] = chi( state[ 3], state[4], C[0] );
    state[ 4] = chi( state[ 4], C[0], C[1] );

    C[0] = state[ 5];
    C[1] = state[ 6];
    state[ 5] = chi( state[ 5], state[6], state[7] );
    state[ 6] = chi( state[ 6], state[7], state[8] );
    state[ 7] = chi( state[ 7], state[8], state[9] );
    state[ 8] = chi( state[ 8], state[9], C[0] );
    state[ 9] = chi( state[ 9], C[0], C[1] );

    C[0] = state[10];
    C[1] = state[11];
    state[10] = chi( state[10], state[11], state[12] );
    state[11] = chi( state[11], state[12], state[13] );
    state[12] = chi( state[12], state[13], state[14] );
    state[13] = chi( state[13], state[14], C[0] );
    state[14] = chi( state[14], C[0], C[1] );

    C[0] = state[15];
    C[1] = state[16];
    state[15] = chi( state[15], state[16], state[17] );
    state[16] = chi( state[16], state[17], state[18] );
    state[17] = chi( state[17], state[18], state[19] );
    state[18] = chi( state[18], state[19], C[0] );
    state[19] = chi( state[19], C[0], C[1] );

    C[0] = state[20];
    C[1] = state[21];
    state[20] = chi( state[20], state[21], state[22] );
    state[21] = chi( state[21], state[22], state[23] );
    state[22] = chi( state[22], state[23], state[24] );
    state[23] = chi( state[23], state[24], C[0] );
    state[24] = chi( state[24], C[0], C[1] );
  }

  for (x = 0; x < 5; ++x)
  {
    C[x] = xor5( state[x], state[x + 5], state[x + 10], state[x + 15], state[x + 20] );
  }

  state[ 0] = state[ 0] ^ ROTL64(C[1], 1) ^ C[4];
  state[ 6] = state[ 6] ^ ROTL64(C[2], 1) ^ C[0];
  state[12] = state[12] ^ ROTL64(C[3], 1) ^ C[1];

  return bswap_64( chi( state[ 0], ROTL64(state[ 6], 44), ROTL64(state[12], 43) ) ^ RC[23] ) <= d_target;
}

// hash length is 256 bits
#if __CUDA_ARCH__ > 500
__global__ __launch_bounds__( TPB52, 1 )
#else
__global__ __launch_bounds__( TPB50, 2 )
#endif
void gpu_mine( uint64_t* solution, int32_t* done, uint64_t cnt, uint32_t threads )
{
  uint64_t thread = blockDim.x * blockIdx.x + threadIdx.x;
  uint64_t nounce = cnt + thread;

#if __CUDA_ARCH__ > 500
  uint64_t step = gridDim.x * blockDim.x;
  uint64_t maxNonce = cnt + threads;
  for( ; nounce < maxNonce; nounce += step )
  {
#else
  if( thread < threads )
  {
#endif

    if( keccak( nounce ) )
    {
      atomicExch( &done[0], thread );
      if( done[0] == thread )
      {
        *solution = nounce;
      }
      return;
    }
  }
}

__host__
void stop_solving()
{
  h_done[0] = -2;
}

__host__
int32_t gcd( int32_t a, int32_t b )
{
  return ( a == 0 ) ? b : gcd( b % a, a );
}

__host__
uint64_t getHashCount()
{
  return cnt;
}
__host__
void resetHashCount()
{
  h_done[0] = 0;

  hipMemcpy( d_done, h_done, sizeof( h_done ), hipMemcpyHostToDevice );
  hipMemset( d_solution, 0xff, 8 );

  printable_hashrate_cnt = 0;
  print_counter = 0;

  ftime( &start );
}

__host__
void send_to_device( uint64_t target, uint64_t* message )
{
  uint64_t C[4], D[5], mid[25];
  C[0] = message[0] ^ message[5] ^ message[10] ^ 0x100000000ull;
  C[1] = message[1] ^ message[6] ^ 0x8000000000000000ull;
  C[2] = message[2] ^ message[7];
  C[3] = message[4] ^ message[9];

  D[0] = ROTL64(C[1], 1) ^ C[3];
  D[1] = ROTL64(C[2], 1) ^ C[0];
  D[2] = ROTL64(message[3], 1) ^ C[1];
  D[3] = ROTL64(C[3], 1) ^ C[2];
  D[4] = ROTL64(C[0], 1) ^ message[3];

  mid[ 0] = message[ 0] ^ D[0];
  mid[ 1] = ROTL64( message[6] ^ D[1], 44 );
  mid[ 2] = ROTL64(D[2], 43);
  mid[ 3] = ROTL64(D[3], 21);
  mid[ 4] = ROTL64(D[4], 14);
  mid[ 5] = ROTL64( message[3] ^ D[3], 28 );
  mid[ 6] = ROTL64( message[9] ^ D[4], 20 );
  mid[ 7] = ROTL64( message[10] ^ D[0] ^ 0x100000000ull, 3 );
  mid[ 8] = ROTL64( 0x8000000000000000ull ^ D[1], 45 );
  mid[ 9] = ROTL64(D[2], 61);
  mid[10] = ROTL64( message[1] ^ D[1],  1 );
  mid[11] = ROTL64( message[7] ^ D[2],  6 );
  mid[12] = ROTL64(D[3], 25);
  mid[13] = ROTL64(D[4],  8);
  mid[14] = ROTL64(D[0], 18);
  mid[15] = ROTL64( message[4] ^ D[4], 27 );
  mid[16] = ROTL64( message[5] ^ D[0], 36 );
  mid[17] = ROTL64(D[1], 10);
  mid[18] = ROTL64(D[2], 15);
  mid[19] = ROTL64(D[3], 56);
  mid[20] = ROTL64( message[2] ^ D[2], 62 );
  mid[21] = ROTL64(D[3], 55);
  mid[22] = ROTL64(D[4], 39);
  mid[23] = ROTL64(D[0], 41);
  mid[24] = ROTL64(D[1],  2);

  hipMemcpyToSymbol(HIP_SYMBOL( d_mid), mid, sizeof( mid ), cuda_device, hipMemcpyHostToDevice);

  hipMemcpyToSymbol(HIP_SYMBOL( d_target), &target, sizeof( target ), cuda_device, hipMemcpyHostToDevice);
}

/**
 * Initializes the global variables by calling the hipGetDeviceProperties().
 */
__host__
void gpu_init()
{
  hipDeviceProp_t device_prop;
  int32_t device_count;

  srand((time(NULL) & 0xFFFF) | (getpid() << 16));

  char config[10];
  FILE * inf;
  inf = fopen( "0xbtc.conf", "r" );
  if( inf )
  {
    fgets( config, 10, inf );
    fclose( inf );
    intensity = atol( strtok( config, " " ) );
    cuda_device = atol( strtok( NULL, " " ) );
  }
  else
  {
    intensity = INTENSITY;
    cuda_device = CUDA_DEVICE;
  }

  hipGetDeviceCount( &device_count );

  if( hipGetDeviceProperties( &device_prop, cuda_device ) != hipSuccess )
  {
    printf( "Problem getting properties for device, exiting...\n" );
    exit( EXIT_FAILURE );
  }

  hipSetDevice( cuda_device );

  if( !gpu_initialized )
  {
    // CPU usage goes _insane_ without this.
    hipDeviceReset();
    hipSetDeviceFlags( hipDeviceScheduleBlockingSync | hipDeviceLmemResizeToMax );
    hipDeviceSetCacheConfig( hipFuncCachePreferL1 );

    hipMalloc( (void**)&d_done, sizeof( int32_t ) );
    hipMalloc( (void**)&d_solution, 32 ); // solution
    hipHostMalloc( (void**)&h_message, 32 );

    (uint32_t&)(init_message[52]) = 014533075101u;
    (uint32_t&)(init_message[56]) = 014132271150u;
    for(int8_t i_rand = 60; i_rand < 84; ++i_rand){
      init_message[i_rand] = (uint8_t)rand() % 256;
    }
    memcpy( solution, &init_message[52], 32 );

    gpu_initialized = true;
  }

  compute_version = device_prop.major * 100 + device_prop.minor * 10;

  // convert from GHz to hertz
  clock_speed = (int32_t)( device_prop.memoryClockRate * 1000 * 1000 );

  //cnt = 0;
  printable_hashrate_cnt = 0;
  print_counter = 0;

  ftime( &start );
  if( new_input ) new_input = false;
}

__host__
void update_mining_inputs( uint64_t target, uint8_t* hash_prefix )
{
  memcpy( init_message, hash_prefix, 52 );
  send_to_device( target, (uint64_t*)init_message );
}

__host__
void gpu_cleanup()
{
  if( !gpu_initialized ) return;

  hipDeviceSynchronize();

  hipFree( d_done );
  hipFree( d_solution );
  hipHostFree( h_message );

  hipDeviceReset();
}

__host__
bool find_message()
{
  uint32_t threads = 1UL << intensity;

  uint32_t tpb;
  dim3 grid;
  if( compute_version > 500 )
  {
    tpb = TPB52;
    grid.x = ( threads + ( NPT*tpb ) - 1 ) / ( NPT*tpb );
  }
  else
  {
    tpb = TPB50;
    grid.x = ( threads + tpb - 1 ) / tpb;
  }
  const dim3 block( tpb );

  gpu_mine <<< grid, block >>> ( d_solution, d_done, cnt, threads );
  // hipError_t cudaerr = hipDeviceSynchronize();
  // if( cudaerr != hipSuccess )
  // {
  //  printf( "kernel launch failed with error %d: \x1b[38;5;196m%s.\x1b[0m\n", cudaerr, hipGetErrorString( cudaerr ) );
  //  exit( EXIT_FAILURE );
  // }

  if( h_done[0] < 0 ) return false;

  cnt += threads;
  printable_hashrate_cnt += threads;

  hipMemcpy( h_done, d_done, sizeof( h_done ), hipMemcpyDeviceToHost );
  if( h_done[0] > 0 )
  {
    hipMemcpy( h_message, d_solution, sizeof( h_message ), hipMemcpyDeviceToHost );
    memcpy( &solution[12], h_message, sizeof( h_message ) );
  }

  ftime( &end );
  double t = (double)((end.time * 1000 + end.millitm) - (start.time * 1000 + start.millitm)) / 1000;

  if( t*10 > print_counter )
  {
    ++print_counter;

    // maybe breaking the control codes into macros is a good idea . . .
    printf( "\x1b[s\x1b[?25l\x1b[2;22f\x1b[38;5;221m%*.2f\x1b[0m\x1b[u\x1b[?25h"
            "\x1b[s\x1b[?25l\x1b[3;36f\x1b[38;5;208m%*" PRIu64 "\x1b[0m\x1b[u\x1b[?25h"
            "\x1b[s\x1b[?25l\x1b[2;75f\x1b[38;5;33m%02u:%02u\x1b[0m\x1b[u\x1b[?25h",
            8, ( (double)printable_hashrate_cnt / t / 1000000 ),
            25, printable_hashrate_cnt,
            ((uint32_t)t/60), ((uint32_t)t%60) );
  }

  return ( h_done[0] > 0 );
}
