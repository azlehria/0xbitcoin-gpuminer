#include "hip/hip_runtime.h"
/*
Author: Mikers
date march 4, 2018 for 0xbitcoin dev

based off of https://github.com/Dunhili/SHA3-gpu-brute-force-cracker/blob/master/sha3.cu

 * Author: Brian Bowden
 * Date: 5/12/14
 *
 * This is the parallel version of SHA-3.
 */

#include "cuda_sha3.h"

int32_t intensity;
int32_t cuda_device;
int32_t clock_speed;
int32_t compute_version;
struct timeb start, end;

uint64_t cnt;
uint64_t printable_hashrate_cnt;
uint64_t print_counter;

bool gpu_initialized;
bool new_input;

uint8_t solution[32] = { 0 };

uint64_t* h_message;
uint8_t init_message[84];

uint64_t* d_solution;

uint8_t* d_challenge;
uint8_t* d_hash_prefix;
__constant__ uint64_t d_mid[25];
__constant__ uint64_t d_target;
__constant__ uint32_t threads;

__device__ __constant__ const uint64_t RC[24] = {
  0x0000000000000001, 0x0000000000008082, 0x800000000000808a,
  0x8000000080008000, 0x000000000000808b, 0x0000000080000001,
  0x8000000080008081, 0x8000000000008009, 0x000000000000008a,
  0x0000000000000088, 0x0000000080008009, 0x000000008000000a,
  0x000000008000808b, 0x800000000000008b, 0x8000000000008089,
  0x8000000000008003, 0x8000000000008002, 0x8000000000000080,
  0x000000000000800a, 0x800000008000000a, 0x8000000080008081,
  0x8000000000008080, 0x0000000080000001, 0x8000000080008008
};

__device__ __forceinline__
uint64_t bswap_64( uint64_t input )
{
  uint64_t output{ 0 };
  asm( "{"
       "  prmt.b32 %0, %3, 0, 0x0123;"
       "  prmt.b32 %1, %2, 0, 0x0123;"
       "}" : "=r"(((uint2&)output).x), "=r"(((uint2&)output).y) : "r"(((uint2&)input).x), "r"(((uint2&)input).y) );
  return output;
}

__device__ __forceinline__
uint64_t xor5( uint64_t a, uint64_t b, uint64_t c, uint64_t d, uint64_t e )
{
  uint64_t output{ 0 };
  asm( "{"
       "  xor.b64 %0, %1, %2;"
       "  xor.b64 %0, %0, %3;"
       "  xor.b64 %0, %0, %4;"
       "  xor.b64 %0, %0, %5;"
       "}" : "+l"(output) : "l"(a), "l"(b), "l"(c), "l"(d), "l"(e) );
  return output;
}

__device__ __forceinline__
uint64_t xor3( uint64_t a, uint64_t b, uint64_t c )
{
  uint64_t output{ 0 };
  asm( "{"
       "  xor.b64 %0, %1, %2;"
       "  xor.b64 %0, %0, %3;"
       "}" : "+l"(output) : "l"(a), "l"(b), "l"(c) );
  return output;
}

__device__ __forceinline__
uint64_t chi( uint64_t a, uint64_t b, uint64_t c )
{
#if __CUDA_ARCH__ >= 500
  uint64_t output{ 0 };
  asm( "{"
       "  lop3.b32 %0, %2, %3, %4, 0xD2;"
       "  lop3.b32 %1, %5, %6, %7, 0xD2;"
       "}" : "=r"(((uint2&)output).x), "=r"(((uint2&)output).y)
           : "r"(((uint2&)a).x), "r"(((uint2&)b).x), "r"(((uint2&)c).x),
             "r"(((uint2&)a).y), "r"(((uint2&)b).y), "r"(((uint2&)c).y) );
  return output;
#else
  return a ^ ((~b) & c);
#endif
}

__device__
bool keccak( uint64_t nounce )
{
  uint64_t state[25], C[5], D[5];

  C[0] = d_mid[ 2] ^ ROTR64(nounce, 20);
  C[1] = d_mid[ 4] ^ ROTL64(nounce, 14);
  state[ 0] = chi( d_mid[ 0], d_mid[ 1], C[ 0] ) ^ RC[0];
  state[ 1] = chi( d_mid[ 1], C[ 0], d_mid[ 3] );
  state[ 2] = chi( C[ 0], d_mid[ 3], C[ 1] );
  state[ 3] = chi( d_mid[ 3], C[ 1], d_mid[ 0] );
  state[ 4] = chi( C[ 1], d_mid[ 0], d_mid[ 1] );

  C[0] = d_mid[ 6] ^ ROTL64(nounce, 20);
  C[1] = d_mid[ 9] ^ ROTR64(nounce,  2);
  state[ 5] = chi( d_mid[ 5], C[ 0], d_mid[7] );
  state[ 6] = chi( C[0], d_mid[ 7], d_mid[8] );
  state[ 7] = chi( d_mid[ 7], d_mid[ 8], C[1] );
  state[ 8] = chi( d_mid[ 8], C[1], d_mid[5] );
  state[ 9] = chi( C[1], d_mid[ 5], C[0] );

  C[0] = d_mid[11] ^ ROTL64(nounce, 7);
  C[1] = d_mid[13] ^ ROTL64(nounce, 8);
  state[10] = chi( d_mid[10], C[0], d_mid[12] );
  state[11] = chi( C[0], d_mid[12], C[1] );
  state[12] = chi( d_mid[12], C[1], d_mid[14] );
  state[13] = chi( C[1], d_mid[14], d_mid[10] );
  state[14] = chi( d_mid[14], d_mid[10], C[0] );

  C[0] = d_mid[15] ^ ROTL64(nounce, 27);
  C[1] = d_mid[18] ^ ROTL64(nounce, 16);
  state[15] = chi( C[0], d_mid[16], d_mid[17] );
  state[16] = chi( d_mid[16], d_mid[17], C[1] );
  state[17] = chi( d_mid[17], C[1], d_mid[19] );
  state[18] = chi( C[1], d_mid[19], C[0] );
  state[19] = chi( d_mid[19], C[0], d_mid[16] );

  C[0] = d_mid[20] ^ ROTR64(nounce,  1);
  C[1] = d_mid[21] ^ ROTR64(nounce,  9);
  C[2] = d_mid[22] ^ ROTR64(nounce, 25);
  state[20] = chi( C[0], C[1], C[2] );
  state[21] = chi( C[1], C[2], d_mid[23] );
  state[22] = chi( C[2], d_mid[23], d_mid[24] );
  state[23] = chi( d_mid[23], d_mid[24], C[0] );
  state[24] = chi( d_mid[24], C[0], C[1] );

#if __CUDA_ARCH__ >= 350
#  pragma unroll
#endif
  for( int32_t i{ 1 }; i < 23; ++i )
  {
    // Theta
    for( uint32_t x{ 0 }; x < 5; ++x )
    {
      C[(x + 6) % 5] = xor5( state[x], state[x + 5], state[x + 10], state[x + 15], state[x + 20] );
    }

#if __CUDA_ARCH__ >= 350
    for( uint32_t x{ 0 }; x < 5; ++x )
    {
			D[x] = ROTL64(C[(x + 2) % 5], 1);
      state[x]      = xor3( state[x]     , D[x], C[x] );
      state[x +  5] = xor3( state[x +  5], D[x], C[x] );
      state[x + 10] = xor3( state[x + 10], D[x], C[x] );
      state[x + 15] = xor3( state[x + 15], D[x], C[x] );
      state[x + 20] = xor3( state[x + 20], D[x], C[x] );
    }
#else
    for( uint32_t x{ 0 }; x < 5; ++x )
    {
      D[x] = ROTL64(C[(x + 2) % 5], 1) ^ C[x];
      state[x]      = state[x]      ^ D[x];
      state[x +  5] = state[x +  5] ^ D[x];
      state[x + 10] = state[x + 10] ^ D[x];
      state[x + 15] = state[x + 15] ^ D[x];
      state[x + 20] = state[x + 20] ^ D[x];
    }
#endif

    // Rho Pi
    C[0] = state[1];
    state[ 1] = ROTR64( state[ 6], 20 );
    state[ 6] = ROTL64( state[ 9], 20 );
    state[ 9] = ROTR64( state[22],  3 );
    state[22] = ROTR64( state[14], 25 );
    state[14] = ROTL64( state[20], 18 );
    state[20] = ROTR64( state[ 2],  2 );
    state[ 2] = ROTR64( state[12], 21 );
    state[12] = ROTL64( state[13], 25 );
    state[13] = ROTL64( state[19],  8 );
    state[19] = ROTR64( state[23],  8 );
    state[23] = ROTR64( state[15], 23 );
    state[15] = ROTL64( state[ 4], 27 );
    state[ 4] = ROTL64( state[24], 14 );
    state[24] = ROTL64( state[21],  2 );
    state[21] = ROTR64( state[ 8],  9 );
    state[ 8] = ROTR64( state[16], 19 );
    state[16] = ROTR64( state[ 5], 28 );
    state[ 5] = ROTL64( state[ 3], 28 );
    state[ 3] = ROTL64( state[18], 21 );
    state[18] = ROTL64( state[17], 15 );
    state[17] = ROTL64( state[11], 10 );
    state[11] = ROTL64( state[ 7],  6 );
    state[ 7] = ROTL64( state[10],  3 );
    state[10] = ROTL64( C[0], 1 );

    // Chi
    for( uint32_t x{ 0 }; x < 25; x += 5 )
    {
      C[0] = state[x];
      C[1] = state[x + 1];
      C[2] = state[x + 2];
      C[3] = state[x + 3];
      C[4] = state[x + 4];
      state[x]     = chi( C[0], C[1], C[2] );
      state[x + 1] = chi( C[1], C[2], C[3] );
      state[x + 2] = chi( C[2], C[3], C[4] );
      state[x + 3] = chi( C[3], C[4], C[0] );
      state[x + 4] = chi( C[4], C[0], C[1] );
    }

    // Iota
    state[0] = state[0] ^ RC[i];
  }

  for( uint32_t x{ 0 }; x < 5; ++x )
  {
    C[(x + 6) % 5 ] = xor5( state[x], state[x + 5], state[x + 10], state[x + 15], state[x + 20] );
  }

  D[0] = ROTL64(C[2], 1);
  D[1] = ROTL64(C[3], 1);
  D[2] = ROTL64(C[4], 1);

  state[ 0] = xor3( state[ 0], D[0], C[0] );
  state[ 6] = ROTR64(xor3( state[ 6], D[1], C[1] ), 20);
  state[12] = ROTR64(xor3( state[12], D[2], C[2] ), 21);

  state[ 0] = chi( state[ 0], state[ 6], state[12] ) ^ RC[23];

  return bswap_64( state[0] ) <= d_target;
}

KERNEL_LAUNCH_PARAMS
void gpu_mine( uint64_t* solution, uint64_t cnt )
{
  uint64_t nounce{ cnt + (blockDim.x * blockIdx.x + threadIdx.x) };

  if( keccak( nounce ) )
  {
    *solution = nounce;
    return;
  }
}

__host__
void stop_solving()
{
  // h_done[0] = -2;
}

__host__
uint64_t getHashCount()
{
  return cnt;
}

__host__
void resetHashCount()
{
  hipSetDevice( cuda_device );

  *h_message = UINT64_MAX;
  hipMemcpy( d_solution, h_message, sizeof( uint64_t ), hipMemcpyHostToDevice );

  printable_hashrate_cnt = 0;
  print_counter = 0;

  ftime( &start );
}

__host__
void send_to_device( uint64_t target, uint64_t* message )
{
  hipSetDevice( cuda_device );

  uint64_t C[4], D[5], mid[25];
  C[0] = message[0] ^ message[5] ^ message[10] ^ 0x100000000ull;
  C[1] = message[1] ^ message[6] ^ 0x8000000000000000ull;
  C[2] = message[2] ^ message[7];
  C[3] = message[4] ^ message[9];

  D[0] = ROTL64(C[1], 1) ^ C[3];
  D[1] = ROTL64(C[2], 1) ^ C[0];
  D[2] = ROTL64(message[3], 1) ^ C[1];
  D[3] = ROTL64(C[3], 1) ^ C[2];
  D[4] = ROTL64(C[0], 1) ^ message[3];

  mid[ 0] = message[ 0] ^ D[0];
  mid[ 1] = ROTL64( message[6] ^ D[1], 44 );
  mid[ 2] = ROTL64(D[2], 43);
  mid[ 3] = ROTL64(D[3], 21);
  mid[ 4] = ROTL64(D[4], 14);
  mid[ 5] = ROTL64( message[3] ^ D[3], 28 );
  mid[ 6] = ROTL64( message[9] ^ D[4], 20 );
  mid[ 7] = ROTL64( message[10] ^ D[0] ^ 0x100000000ull, 3 );
  mid[ 8] = ROTL64( 0x8000000000000000ull ^ D[1], 45 );
  mid[ 9] = ROTL64(D[2], 61);
  mid[10] = ROTL64( message[1] ^ D[1],  1 );
  mid[11] = ROTL64( message[7] ^ D[2],  6 );
  mid[12] = ROTL64(D[3], 25);
  mid[13] = ROTL64(D[4],  8);
  mid[14] = ROTL64(D[0], 18);
  mid[15] = ROTL64( message[4] ^ D[4], 27 );
  mid[16] = ROTL64( message[5] ^ D[0], 36 );
  mid[17] = ROTL64(D[1], 10);
  mid[18] = ROTL64(D[2], 15);
  mid[19] = ROTL64(D[3], 56);
  mid[20] = ROTL64( message[2] ^ D[2], 62 );
  mid[21] = ROTL64(D[3], 55);
  mid[22] = ROTL64(D[4], 39);
  mid[23] = ROTL64(D[0], 41);
  mid[24] = ROTL64(D[1],  2);

  hipMemcpyToSymbol(HIP_SYMBOL( d_mid), mid, sizeof( mid ), 0, hipMemcpyHostToDevice);

  hipMemcpyToSymbol(HIP_SYMBOL( d_target), &target, sizeof( target ), 0, hipMemcpyHostToDevice);
}

/**
 * Initializes the global variables by calling the hipGetDeviceProperties().
 */
__host__
void gpu_init()
{
  hipDeviceProp_t device_prop;
  int32_t device_count;

  char config[10];
  FILE * inf;
  inf = fopen( "0xbtc.conf", "r" );
  if( inf )
  {
    fgets( config, 10, inf );
    fclose( inf );
    intensity = atol( strtok( config, " " ) );
    cuda_device = atol( strtok( NULL, " " ) );
  }
  else
  {
    intensity = INTENSITY;
    cuda_device = CUDA_DEVICE;
  }

  hipGetDeviceCount( &device_count );

  if( hipGetDeviceProperties( &device_prop, cuda_device ) != hipSuccess )
  {
    printf( "Problem getting properties for device, exiting...\n" );
    exit( EXIT_FAILURE );
  }

  hipSetDevice( cuda_device );

  if( !gpu_initialized )
  {
    // CPU usage goes _insane_ without this.
    hipDeviceReset();
    hipSetDeviceFlags( hipDeviceScheduleBlockingSync | hipDeviceLmemResizeToMax );
    hipDeviceSetCacheConfig( hipFuncCachePreferL1 );

    hipMalloc( (void**)&d_solution, sizeof( uint64_t ) ); // solution
    hipHostMalloc( (void**)&h_message, sizeof( uint64_t ) );

    (uint32_t&)(init_message[52]) = 014533075101u;
    (uint32_t&)(init_message[56]) = 014132271150u;

    srand((time(NULL) & 0xFFFF) | (getpid() << 16));
    for(int8_t i_rand{ 60 }; i_rand < 84; ++i_rand){
      init_message[i_rand] = (uint8_t)rand() % 256;
    }
    memcpy( solution, &init_message[52], 32 );

    uint32_t h_threads{ 1u << intensity };
    hipMemcpyToSymbol(HIP_SYMBOL( threads), &h_threads, sizeof( h_threads ), 0, hipMemcpyHostToDevice );

    gpu_initialized = true;
  }

  compute_version = device_prop.major * 100 + device_prop.minor * 10;

  // convert from GHz to hertz
  clock_speed = (int32_t)( device_prop.memoryClockRate * 1000 * 1000 );

  //cnt = 0;
  printable_hashrate_cnt = 0;
  print_counter = 0;

  ftime( &start );
  if( new_input ) new_input = false;
}

__host__
void update_mining_inputs( uint64_t target, uint8_t* hash_prefix )
{
  memcpy( init_message, hash_prefix, 52 );
  send_to_device( target, (uint64_t*)init_message );
}

__host__
void gpu_cleanup()
{
  hipSetDevice( cuda_device );

  hipDeviceSynchronize();

  hipFree( d_solution );
  hipHostFree( h_message );

  hipDeviceReset();
}

__host__
bool find_message()
{
  hipSetDevice( cuda_device );

  uint32_t threads{ 1u << intensity };

  uint32_t tpb{ compute_version > 500 ? TPB50 : TPB35 };
  dim3 grid{ (threads + tpb - 1) / tpb };
  dim3 block{ tpb };

  gpu_mine <<< grid, block >>> ( d_solution, cnt );
  // hipError_t cudaerr = hipDeviceSynchronize();
  // if( cudaerr != hipSuccess )
  // {
  //  printf( "kernel launch failed with error %d: \x1b[38;5;196m%s.\x1b[0m\n", cudaerr, hipGetErrorString( cudaerr ) );
  //  exit( EXIT_FAILURE );
  // }

  cnt += threads;
  printable_hashrate_cnt += threads;

  hipMemcpy( h_message, d_solution, sizeof( uint64_t ), hipMemcpyDeviceToHost );
  if( *h_message != UINT64_MAX )
    memcpy( &solution[12], h_message, sizeof( uint64_t ) );

  ftime( &end );
  double t{ (double)((end.time * 1000 + end.millitm) - (start.time * 1000 + start.millitm)) / 1000 };

  if( t*10 > print_counter )
  {
    ++print_counter;

    // maybe breaking the control codes into macros is a good idea . . .
    printf( "\x1b[s\x1b[?25l\x1b[2;22f\x1b[38;5;221m%*.2f\x1b[0m\x1b[u\x1b[?25h"
            "\x1b[s\x1b[?25l\x1b[3;36f\x1b[38;5;208m%*" PRIu64 "\x1b[0m\x1b[u\x1b[?25h"
            "\x1b[s\x1b[?25l\x1b[2;75f\x1b[38;5;33m%02u:%02u\x1b[0m\x1b[u\x1b[?25h",
            8, ( (double)printable_hashrate_cnt / t / 1000000 ),
            25, printable_hashrate_cnt,
            ((uint32_t)t/60), ((uint32_t)t%60) );
  }

  return ( *h_message != UINT64_MAX );
  // return ( h_done[0] >= 0 );
}
