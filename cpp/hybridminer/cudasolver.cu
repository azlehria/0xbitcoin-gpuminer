#include "cudasolver.h"
#include "sha3.h"

#include <assert.h>
#include <sstream>
#include <iomanip>
#include <stdio.h>
#include <iostream>
#include <string.h>

//we will need this!
#include "cuda_sha3.cu"

extern int h_done[1];
extern unsigned char* h_message;

static uint8_t fromAscii( uint8_t c )
{
  if( c >= '0' && c <= '9' )
    return ( c - '0' );
  if( c >= 'a' && c <= 'f' )
    return ( c - 'a' + 10 );
  if( c >= 'A' && c <= 'F' )
    return ( c - 'A' + 10 );

#if defined(__EXCEPTIONS) || defined(DEBUG)
  throw std::runtime_error( "invalid character" );
#else
  return 0xff;
#endif
}

static uint8_t ascii_r( uint8_t a, uint8_t b )
{
  return fromAscii( a ) * 16 + fromAscii( b );
}

static void HexToBytes( std::string const& hex, uint8_t bytes[] )
{
  for( std::string::size_type i = 0, j = 0; i < hex.length(); i += 2, ++j )
  {
    bytes[j] = ascii_r( hex[i], hex[i + 1] );
  }
}

// --------------------------------------------------------------------

// static
std::atomic<uint32_t> CUDASolver::hashes( 0u ); // statistics only

CUDASolver::CUDASolver() noexcept :
m_address( ADDRESS_LENGTH ),
m_challenge( UINT256_LENGTH ),
m_target( UINT256_LENGTH ),
m_target_tmp( UINT256_LENGTH ),
m_buffer( ADDRESS_LENGTH + 2 * UINT256_LENGTH ),
m_buffer_tmp( ADDRESS_LENGTH + 2 * UINT256_LENGTH ), //this has something to do with updateBuffer
m_buffer_ready( false ),
m_target_ready( false ),
m_updated_gpu_inputs( false )
{
}

void CUDASolver::setAddress( std::string const& addr )
{
  std::cout << "Setting cuda addr " << std::endl;

  assert( addr.length() == ( ADDRESS_LENGTH * 2 + 2 ) );
  hexToBytes( addr, m_address );
  //updateBuffer();

  m_updated_gpu_inputs = true;
  updateGPULoop();
}

void CUDASolver::setChallenge( std::string const& chal )
{
  //std::cout << "Setting cuda chal " << std::endl;

  s_challenge = chal;

  assert( chal.length() == ( UINT256_LENGTH * 2 + 2 ) );
  hexToBytes( chal, m_challenge );
  //updateBuffer();
  m_updated_gpu_inputs = true;
  updateGPULoop();
}

void CUDASolver::setTarget( std::string const& target )
{
  std::cout << "Setting cuda tar " << target << std::endl;

  assert( target.length() <= ( UINT256_LENGTH * 2 + 2 ) );
  std::string const t( static_cast<std::string::size_type>( UINT256_LENGTH * 2 + 2 ) - target.length(), '0' );

  s_target = target;

  // Double-buffer system, the trySolution() function will be blocked
  //  only when a change occurs.
  {
    std::lock_guard<std::mutex> g( m_target_mutex );
    hexToBytes( "0x" + t + target.substr( 2 ), m_target_tmp );
  }
  m_target_ready = true;

  m_updated_gpu_inputs = true;
  updateGPULoop();
}

bool CUDASolver::requiresRestart()
{
  return m_updated_gpu_inputs;
}

//This will restart the miner if needed
void CUDASolver::updateGPULoop()
{
  if( m_updated_gpu_inputs
      && m_target_ready
      && m_challenge.size() > 0
      && m_address.size() > 0 )
  {
    m_updated_gpu_inputs = false;

    //printf( "Target input:\n" );

    if( s_target.length() < 66 )
    {
      std::string zeros = std::string( 66 - s_target.length(), '0' );
      std::string s = "0x" + zeros + s_target.substr( 2, s_target.length() );
      s_target = s;
    }

    unsigned char target_input[64];
    bytes_t target_bytes( 32 );

    hexToBytes( s_target, target_bytes );

    for( int i = 0; i < 32; i++ )
    {
      target_input[i] = (unsigned char)target_bytes[i];
      //printf( "%02x", (unsigned char)target_input[i] );
    }

    unsigned char hash_prefix[52];
    std::string clean_challenge = s_challenge;
    bytes_t challenge_bytes( 32 );

    hexToBytes( clean_challenge, challenge_bytes );

    for( int i = 0; i < 32; i++ )
    {
      hash_prefix[i] = (unsigned char)challenge_bytes[i];
    }
    for( int i = 0; i < 20; i++ )
    {
      hash_prefix[i + 32] = (unsigned char)m_address[i];
    }

    //printf( "Challenge+Address:\n" );
    //for( int i = 0; i < 52; i++ )
    //{
    //  printf( "%02x", (unsigned char)hash_prefix[i] );
    //}
    //printf( "\n/prefix\n" );

    //printf( "Updating mining inputs\n" );
    update_mining_inputs( target_input, hash_prefix );
    stop_solving();
  }
}

// Buffer order: 1-challenge 2-ethAddress 3-solution
/*
void CUDASolver::updateBuffer()
{
  // The idea is to have a double-buffer system in order not to try
  //  to acquire a lock on each hash() loop
  {
    std::lock_guard<std::mutex> g(m_buffer_mutex);
    std::copy(m_challenge.cbegin(), m_challenge.cend(), m_buffer_tmp.begin());
    std::copy(m_address.cbegin(), m_address.cend(), m_buffer_tmp.begin() + m_challenge.size());
  }
  m_buffer_ready = true;
}*/

//call the sha3.cu init func
void CUDASolver::init()
{
  std::cout << "CUDA initializing ..." << std::endl;
  gpu_init();
}

void CUDASolver::stopFinding()
{
  //std::cout << "CUDA has stopped hashing for now." << std::endl;

  //set h_done[0] = 1
  stop_solving();
}

CUDASolver::bytes_t CUDASolver::findSolution()
{
  m_updated_gpu_inputs = false;

  //std::cout << "CUDA is trying to find a solution :)" << std::endl;

  // What are these even here for?
  //hipEventCreate( &start );
  //hipEventCreate( &stop );

  //printf( "Target input:\n" );

  if( s_target.length() < 66 )
  {
    std::string zeros = std::string( 66 - s_target.length(), '0' );
    std::string s = "0x" + zeros + s_target.substr( 2, s_target.length() );
    s_target = s;
  }

  unsigned char  target_input[64];
  bytes_t target_bytes( 32 );

  hexToBytes( s_target, target_bytes );

  for( int i = 0; i < 32; i++ )
  {
    target_input[i] = (unsigned char)target_bytes[i];
    //printf( "%02x", (unsigned char)target_input[i] );
  }

  unsigned   char  hash_prefix[52];
  std::string clean_challenge = s_challenge;
  bytes_t challenge_bytes( 32 );

  hexToBytes( clean_challenge, challenge_bytes );

  for( int i = 0; i < 32; i++ )
  {
    hash_prefix[i] = (unsigned char)challenge_bytes[i];
  }
  for( int i = 0; i < 20; i++ )
  {
    hash_prefix[i + 32] = (unsigned char)m_address[i];
  }

  //printf( "\nChallenge+Address:\n" );
  //for( int i = 0; i < 52; i++ )
  //{
  //  printf( "%02x", (unsigned char)hash_prefix[i] );
  //}
  ////printf( "\n/prefix\n" );

  CUDASolver::bytes_t byte_solution( 32 );
  h_done[0] = 0;

  do
  {
    //hipDeviceReset();
    hipSetDeviceFlags( hipDeviceScheduleBlockingSync );

    if( !find_message( target_input, hash_prefix ) )
      continue;
    //here
    for( int i = 0; i < 32; i++ )
    {
      byte_solution[i] = (uint8_t)h_message[i + 52];

      //cout << (uint8_t)s_solution[i];
    }
  } while( !h_done[0] );
  gpu_cleanup();

  // What are these even here for?
  //hipEventDestroy( start );
  //hipEventDestroy( stop );
  printf( "cpu " );
  //for( int i = 0; i < 52; i++ )
  //  printf( "%02x", hash_prefix[i] );
  //for( int i = 0; i < 32; i++ )
  //  printf( "%02x", h_message[i] );
  //bytes_t temp( 84 ), out( 32 );
  //for( int i = 0; i < 52; i++ )
  //  temp[i] = hash_prefix[i];
  //for( int i = 0; i < 32; i++ )
  //  temp[i + 52] = byte_solution[i];
  //hash( temp, out );
  //for( int i = 0; i < 32; i++ )
  //  printf( "%02x", out[i] );
  //printf( "\n" );
  return byte_solution;
}

std::string CUDASolver::hexStr( char* data, int len )
{
  std::stringstream ss;
  ss << std::hex;
  for( int i = 0; i < len; ++i )
    ss << std::setw( 2 ) << std::setfill( '0' ) << (int)data[i];
  return ss.str();
}

// static
void CUDASolver::hexToBytes( std::string const& hex, bytes_t& bytes )
{
  /*
      cout << "hex to bytes: " << hex << "\n";
      cout << bytes.size()  << "\n";
      cout << hex.length()  << "\n";
  */
  assert( hex.length() % 2 == 0 );
  assert( bytes.size() == ( hex.length() / 2 - 1 ) );
  HexToBytes( hex.substr( 2 ), &bytes[0] );
}

// static
std::string CUDASolver::bytesToString( bytes_t const& buffer )
{
  static const char table[] = "0123456789ABCDEF";
  std::string output;
  output.reserve( buffer.size() * 2 + 1 );

  for( unsigned i = 0; i < buffer.size(); ++i )
    output += table[buffer[i]];

  return output;
}

// static
bool CUDASolver::lte( bytes_t const& left, bytes_t const& right )
{
  assert( left.size() == right.size() );

  for( unsigned i = 0; i < left.size(); ++i )
  {
    if( left[i] == right[i] )
      continue;
    if( left[i] > right[i] )
      return false;
    return true;
  }
  return true;
}

void CUDASolver::hash( bytes_t const& solution, bytes_t& digest )
{
  if( m_buffer_ready )
  {
    std::lock_guard<std::mutex> g( m_buffer_mutex );
    m_buffer.swap( m_buffer_tmp );
    m_buffer_ready = false;
  }

  std::copy( solution.cbegin(), solution.cend(), m_buffer.begin() + m_challenge.size() + m_address.size() );
  keccak_256( &digest[0], digest.size(), &m_buffer[0], m_buffer.size() );
}

bool CUDASolver::trySolution( bytes_t const& solution )
{
  bytes_t digest( UINT256_LENGTH );
  hash( solution, digest );

  if( m_target_ready )
  {
    std::lock_guard<std::mutex> g( m_target_mutex );
    m_target.swap( m_target_tmp );
    m_target_ready = false;
  }

  return lte( digest, m_target );
}
